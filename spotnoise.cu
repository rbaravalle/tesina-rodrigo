#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define NX 256
#define NY 256

#define tamanio (NX*NY)


// Cuda code

// Complex multiplication
static __device__ __host__ inline hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b)
{
    hipfftComplex c;

    c.x = (a.x * b.x - a.y * b.y) * (1.0f / (float)(tamanio));
    c.y = (a.x * b.y + a.y * b.x) * (1.0f / (float)(tamanio));
    return c;
}

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(hipfftComplex* a, hipfftComplex* b, int size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
        a[i] = ComplexMul(a[i], b[i]);  
} 

void spotNoise(int* spotParam, GLubyte* result)
{


	hipfftHandle plan, planNoise;

	hipfftReal * texturaCuda = (hipfftReal*)malloc(sizeof(hipfftReal)*tamanio);
	hipfftReal * noiseCuda =   (hipfftReal*)malloc(sizeof(hipfftReal)*tamanio);

	int i;
 	for(i = 0; i<tamanio; i++)
	{
			texturaCuda[i] = (hipfftReal)spotParam[i];
			
			int random = (rand() % 256 );
					
			noiseCuda[i] = (hipfftReal)random;
			
	}

	hipfftReal *idata, *idataNoise;
	hipfftComplex *odata, *odataNoise;


	hipMalloc((void**)&idata, sizeof(hipfftReal)*tamanio);
	hipMalloc((void**)&idataNoise, sizeof(hipfftReal)*tamanio);

	hipMalloc((void**)&odata, sizeof(hipfftComplex)*tamanio);
	hipMalloc((void**)&odataNoise, sizeof(hipfftComplex)*tamanio);

	hipfftResult res = hipfftPlan2d(&plan, NX, NY, HIPFFT_R2C);

	// copio al device la imagen
	hipMemcpy(idata, texturaCuda, sizeof(hipfftReal)*tamanio,
                              hipMemcpyHostToDevice);

	hipfftExecR2C( plan, idata, odata );

	hipfftPlan2d(&planNoise, NX, NY, HIPFFT_R2C);

	// copio al device el ruido
	hipMemcpy(idataNoise, noiseCuda, sizeof(hipfftReal)*tamanio,
                              hipMemcpyHostToDevice);

	hipfftExecR2C( planNoise, idataNoise, odataNoise );


	// multiplico en el dominio de las frecuencias (En la GPU)
	// un grid, un block, 256 threads
	ComplexPointwiseMulAndScale<<<32, 256>>>(odata, odataNoise, tamanio);
	// en odata, la transformada que queremos

	hipfftHandle planInv;
	hipfftPlan2d(&planInv, NX, NY, HIPFFT_C2R);

	hipfftExecC2R( planInv, odata, idata );

	hipMemcpy(texturaCuda, idata, sizeof(hipfftReal)*tamanio,
                              hipMemcpyDeviceToHost);


	
 	for(i = 0; i<tamanio; i++)
	{	
		result[i] = (int)(texturaCuda[i]/tamanio);			
	}


	/* Destroy the CUFFT plan. */
	hipfftDestroy(plan);
	hipfftDestroy(planInv);
	hipFree(idata); hipFree(odata);
	hipFree(idataNoise); hipFree(odataNoise);

}